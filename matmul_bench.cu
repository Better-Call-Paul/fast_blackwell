#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <cuda/barrier>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <ctime>
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_bf16.h>
#include <cassert>
#include <unistd.h>
#include <cmath>
#include "kernel_runners.cuh"

#include "cuda_common.cuh"

#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

void cudaCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
        exit(1);
    }
}

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

__global__ void warmupKernel()
{
    __shared__ int s[100];
    s[0] += s[1];
}

std::default_random_engine generator(69);
int yo = 0;
void randomize_matrix(__hip_bfloat16 *mat, int N) 
{
  std::normal_distribution<float> distribution(0, 1);
  for (int i = 0; i < N; i++) 
  {
    mat[i] = distribution(generator);
  }
  ++yo;
}

bool verify_matrix(__hip_bfloat16 *matRef, __hip_bfloat16 *matOut, int N) 
{
  double diff = 0.0;
  int i;
  for (i = 0; i < N; i++) 
  {
    int r = i / 8192, c = i % 8192;
    int it = c*8192+r;
    diff = std::fabs(__bfloat162float(matRef[i]) - __bfloat162float(matOut[i]));
    if (diff > 0.1) 
    {
      printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n",
      __bfloat162float(matRef[i]), __bfloat162float(matOut[i]), diff, i);
      return false;
    }
  }
  return true;
}

hipblasHandle_t cublas_handle;
void runCublasGemmBF16(int M, int N, int K, __hip_bfloat16 *A, __hip_bfloat16 *B, __hip_bfloat16 *C)
{
  float alpha = 1, beta = 0;
  // C(column major) = A(row major) * B(column major)
  hipblasStatus_t status = hipblasGemmEx(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, N, K, &alpha, A, HIP_R_16BF,
    N, B, HIP_R_16BF, K, &beta, C, HIP_R_16BF, N, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);

  if (status != HIPBLAS_STATUS_SUCCESS) 
  {
    std::cout << "CUBLAS error: " << status << std::endl;
    exit(1);
  }
}

int main()
{
  warmupKernel<<<1024, 1024>>>();

  hipblasCreate(&cublas_handle);
  float elapsed_time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int max_size = 8192;
  int M, N, K;
  M = N = K = max_size;

  __hip_bfloat16 *A = nullptr, *B = nullptr, *C = nullptr, *C_ref = nullptr;
  __hip_bfloat16 *dA = nullptr, *dB = nullptr, *dC = nullptr, *dC_ref = nullptr;
  int *device_buffer = nullptr, *d_device_buffer = nullptr;

  A     = (__hip_bfloat16 *)malloc(sizeof(__hip_bfloat16) * M * K);
  B     = (__hip_bfloat16 *)malloc(sizeof(__hip_bfloat16) * K * N);
  C     = (__hip_bfloat16 *)malloc(sizeof(__hip_bfloat16) * M * N);
  C_ref = (__hip_bfloat16 *)malloc(sizeof(__hip_bfloat16) * M * N);

  device_buffer = (int *)malloc(sizeof(int) * max_size * 128);
  cudaCheck(hipMalloc((void**)&d_device_buffer, sizeof(int) * max_size * 128));

  randomize_matrix(A, M * K);
  randomize_matrix(B, K * N);
  randomize_matrix(C, M * N);

  cudaCheck(hipMalloc((void **)&dA, sizeof(__hip_bfloat16) * M * K));
  cudaCheck(hipMalloc((void **)&dB, sizeof(__hip_bfloat16) * K * N));
  cudaCheck(hipMalloc((void **)&dC, sizeof(__hip_bfloat16) * M * N));
  cudaCheck(hipMalloc((void **)&dC_ref, sizeof(__hip_bfloat16) * M * N));
  
  cudaCheck(hipMemcpy(dA, A, sizeof(__hip_bfloat16) * M * K, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dB, B, sizeof(__hip_bfloat16) * K * N, hipMemcpyHostToDevice));

  int repeat_count = 5;
  bool run_verification = false;

  for ( int kernel_num : {0, 1, 2, 3, 4, 5})
  {
    sleep(5);
    std::cout << "KERNEL: " << kernel_num << "\n";

    if (run_verification)
    {
      memset(C, 0, sizeof(__hip_bfloat16) * M * N);
      cudaCheck(hipMemcpy(dC, C, sizeof(__hip_bfloat16) * M * N, hipMemcpyHostToDevice));
      cudaCheck(hipMemcpy(dC_ref, C, sizeof(__hip_bfloat16) * M * N, hipMemcpyHostToDevice));
      
      memset(device_buffer, ~0, sizeof(int) * max_size * 128);
      cudaCheck(hipMemcpy(d_device_buffer, device_buffer, sizeof(int) *max_size, hipMemcpyHostToDevice));

      runCublasGemmBF16(M, N, K, dA, dB, dC_ref);

      // run_kernel(kernel_num, M, N, K, dA, dB, dC);

      cudaCheck(hipDeviceSynchronize());
      cudaCheck(hipGetLastError());
      hipMemcpy(C, dC, sizeof(__hip_bfloat16) * M * N, hipMemcpyDeviceToHost);
      hipMemcpy(C_ref, dC_ref, sizeof(__hip_bfloat16) * M * N, hipMemcpyDeviceToHost);

      if (kernel_num > 1 && !verify_matrix(C_ref, C, M * N))
      {
        std::cout << "~~~~~~~~~~~~~~~~ Failed to pass the correctness verification against cuBLAS. ~~~~~~~~~~~~~~~~\n";
        printf("%f\n", __bfloat162float(C_ref[M]));
      }

      // Retrieve Device Buffer Datapoints

      hipMemcpy(device_buffer, d_device_buffer, sizeof(int) * max_size * 8, hipMemcpyDeviceToHost);

      int i = 0;
      long sumLoad = 0, cntLoad = 0;
      long sumCompute = 0, cntCompute = 0;
      long sumStore = 0, cntStore = 0;
      int times = 0;

      while (device_buffer[i] != 0)
      {
        sumLoad += device_buffer[i], cntLoad += device_buffer[i + 1];
        sumCompute += device_buffer[i + 2], cntCompute += device_buffer[i + 3];
        sumStore += device_buffer[i + 4], cntStore += device_buffer[i + 5];
        i += 6;
        times++;
      }

      if (times > 0)
      {
        printf("Load: %f, Compute: %f,  Store: %f, Datapoints: %d\n", (sumLoad + .0) / cntLoad, (sumCompute + .0) / cntCompute, (sumStore + .0) / cntStore, times);
      }
    
    }

    hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
    // dummy launch
    runCublasGemmBF16(M, N, K, dA, dB, dC_ref);

    hipblasSetStream(cublas_handle, 0);
    hipEventRecord(start, 0);

    for (int j = 0; j < repeat_count; ++j)
    {
        runCublasGemmBF16(M, N, K, dA, dB, dC_ref);
    }
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    long FLOPS = (2LL * M) * (N * K);

    printf(
        "Average elapsed time: (%7.6f) s, performance: (%9.3f) PFLOPS. size: (%ld).\n\n",
        elapsed_time * 1e-3 / repeat_count,
        (repeat_count * FLOPS * 1e-12) / elapsed_time,
        M
    );
        
  }

  free(A);
  free(B);
  free(C);
  free(C_ref);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipFree(dC_ref);

  return 0;
}
